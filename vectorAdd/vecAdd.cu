#include "hip/hip_runtime.h"
#include "vecAdd.h"

__global__ void vectorAddCUDA(REAL_TYPE *A, REAL_TYPE *B, REAL_TYPE *C, int32_t nelem) {
  size_t tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < nelem) {
    C[tid] = A[tid] + B[tid];
  }
}

extern "C" {
void vecAdd(REAL_TYPE *A, REAL_TYPE *B, REAL_TYPE *C, int32_t lo, int32_t hi, int32_t nelem) {
  REAL_TYPE *dA, *dB, *dC;
  int32_t work = hi-lo+1;
  hipSetDevice(dev_num);
  hipMalloc(&dA, sizeof(REAL_TYPE) * work);
  hipMalloc(&dB, sizeof(REAL_TYPE) * work);
  hipMalloc(&dC, sizeof(REAL_TYPE) * work);

  hipMemcpy(dA, A + lo, sizeof(REAL_TYPE) * work, hipMemcpyHostToDevice);
  hipMemcpy(dB, B + lo, sizeof(REAL_TYPE) * work, hipMemcpyHostToDevice);
  dim3 block = {256, 1, 1};
  dim3 grid = {(work / block.x) + (work % block.x ? 1 : 0), 1, 1};
  vectorAddCUDA<<<grid, block>>>(dA, dB, dC, work);
  hipMemcpy(C + lo, dC, sizeof(REAL_TYPE) * work, hipMemcpyDeviceToHost);
}
}
